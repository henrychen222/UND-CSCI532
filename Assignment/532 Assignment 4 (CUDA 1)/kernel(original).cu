#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <time.h>

const int srcHeight = 32;
const int srcWidth = 32;
const int maskHeight = 4;
const int maskWidth = 4;
const int dstHeight = srcHeight - maskHeight + 1;
const int dstWidth = srcWidth - maskWidth + 1;

const int BLOCK_DIM_X = 32;
const int BLOCK_DIM_Y = 32;

__constant__ float devMask[maskHeight*maskWidth];

__global__ void convKernel(int maskW, int maskH, int srcW, int srcH, int dstW, int dstH, float * src, float * dst)
{
    const int tx = blockDim.x * blockIdx.x + threadIdx.x;
	const int ty = blockDim.y * blockIdx.y + threadIdx.y;

	//coordination outside the range, drop them
	if (tx < dstW && ty < dstH)
	{
		dst[ty*dstW + tx] = 0.0;
		for (int cy = 0; cy < maskH; cy++)
		{
			for (int cx = 0; cx < maskW; cx++)
			{
				dst[ty*dstW + tx] += devMask[cy*maskW + cx] * src[ty*srcW + tx];
			}
		}
	}

	return;
}

void printMatrix(const char * name, float * matrix, int height, int width)
{
	printf("%s\n", name);
	for (int idx = 0; idx < height*width; ++idx)
	{
		printf("%4.1f ", matrix[idx]);
		if (idx % width == width - 1)
		{
			printf("\n");
		}
	}
	printf("\n");
}

int main()
{
	//list and select device
	hipDeviceProp_t prop;
	int devCount = 0;
	hipError_t hipError_t = hipGetDeviceCount(&devCount);
	for (int i = 0; i < devCount; ++i)
	{
		hipError_t = hipGetDeviceProperties(&prop, i);
		printf("Cuda Device %d/%d:\n", i, devCount);
		printf("Cuda Device Name: %s\n", prop.name);
		printf("Global Memory: %.2lf MB\n", prop.totalGlobalMem / 1024.0 / 1024.0);
		printf("Shared Memory Per Block: %.2lf KB\n", prop.sharedMemPerBlock / 1024.0);
		printf("Register Per Block: %d\n", prop.regsPerBlock);
		printf("Max Thread Per Block: %d\n", prop.maxThreadsPerBlock);
		printf("Max Size of each dim of block xyz[%d %d %d]\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
		printf("Compute Capability: %d.%d\n", prop.major, prop.minor);
		
	}
	int selectedDev = 0;
	while (1)
	{
		printf("Select a device by device ID: ");
		scanf("%d", &selectedDev);
		if (hipSuccess != hipSetDevice(selectedDev))
		{
			printf("\nInvalid input, try again!\n");
		}
		else
		{
			printf("Device #%d is now in use!\n", selectedDev);
			break;
		}
	}

	//allocate host memory
	float * srcMatrix = (float *)calloc(srcHeight*srcWidth, sizeof(float));
	float * dstMatrix = (float *)calloc(dstHeight*dstWidth, sizeof(float));
	float * maskMatrix = (float *)calloc(maskHeight*maskWidth, sizeof(float));

	for (int idx = 0; idx < srcHeight*srcWidth; ++idx)
	{
		//srand((unsigned)time(NULL) + idx);
		srcMatrix[idx] = 1.1;//(rand() % 100) / 100.0;
	}

	printMatrix("srcMatrix", srcMatrix, srcHeight, srcWidth);

	for (int idx = 0; idx < maskHeight*maskWidth; ++idx)
	{
		//srand((unsigned)time(NULL) + idx);
		maskMatrix[idx] = 1.0;//(rand() % 100) / 100.0;
	}

    //allocate device memory
	float * devSrc;
	float * devDst;

	hipMalloc((float **)&devSrc, srcHeight*srcWidth * sizeof(float));
	hipMalloc((float **)&devDst, dstHeight*dstWidth * sizeof(float));

	hipMemcpyToSymbol(HIP_SYMBOL(devMask), maskMatrix, maskHeight*maskWidth * sizeof(float));
	hipMemcpy(devSrc, srcMatrix, srcHeight*srcWidth * sizeof(float), hipMemcpyHostToDevice);

	//call kernel to do parallel compute
	dim3 block(BLOCK_DIM_X, BLOCK_DIM_Y);
	dim3 grid((srcWidth + block.x - 1) / block.x, (srcHeight + block.y - 1) / block.y);
	convKernel << <grid, block >> > (maskWidth, maskHeight, srcWidth, srcHeight, dstWidth, dstHeight, devSrc, devDst);
	hipDeviceSynchronize();

	hipMemcpy(dstMatrix, devDst, dstHeight*dstWidth * sizeof(float), hipMemcpyDeviceToHost);

	printMatrix("dstMatrix", dstMatrix, dstHeight, dstWidth);

	hipFree(devDst);
	hipFree(devSrc);

	free(srcMatrix);
	free(dstMatrix);
	free(maskMatrix);

    return 0;
}
