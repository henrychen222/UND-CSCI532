
extern "C++" {
#include "stdlib.h"
#include "stdio.h"
}

#include <hip/hip_runtime.h>


void print_array(const char *name, float *array, int array_length) {
    printf("%s: [", name);
    for (int i = 0; i < array_length; i++) printf(" %5.2f", array[i]);
    printf(" ]\n");
}

__global__ void gpu__vector_add(float *v1, float *v2, float *sum) {
    int position = (blockDim.y * threadIdx.x) + threadIdx.y;

    sum[position] = v1[position] + v2[position];
}

int main(int n_arguments, char **arguments) {
    hipSetDevice( 0 );

    size_t free_memory;
    size_t total_memory;
    hipMemGetInfo(&free_memory, &total_memory);
    printf("free memory: %u, total memory: %u (before initialize)\n", (unsigned int)free_memory, (unsigned int)total_memory);


    int array_length = atoi(arguments[1]);

    int number_threads_x = 512 / 4, number_threads_y = 4;
    dim3 blockDimensions(number_threads_x, number_threads_y, 1);

    float *cpu__v1;
    float *cpu__v2;
    float *cpu__sum;

    /**
     *  Allocate the arrays locally
     */
    cpu__v1 = (float*)malloc(array_length * sizeof(float));
    cpu__v2 = (float*)malloc(array_length * sizeof(float));
    cpu__sum = (float*)malloc(array_length * sizeof(float));
//    memset(cpu__v1,  0, array_length * sizeof(float));
//    memset(cpu__v2,  0, array_length * sizeof(float));
//    memset(cpu__sum, 0, array_length * sizeof(float));

    float *gpu__v1;
    float *gpu__v2;
    float *gpu__sum;

    /**
     *  Allocate the memory on the GPU
     */
    hipMalloc((void**) &gpu__v1,  array_length * sizeof(float));
    hipMalloc((void**) &gpu__v2,  array_length * sizeof(float));
    hipMalloc((void**) &gpu__sum, array_length * sizeof(float));

//    cudaMemset(gpu__v1,  0, array_length * sizeof(float));
//    cudaMemset(gpu__v2,  0, array_length * sizeof(float));
//    cudaMemset(gpu__sum, 0, array_length * sizeof(float));

    hipMemGetInfo(&free_memory, &total_memory);
    printf("free memory: %u, total memory: %u (after mallocs)\n", (unsigned int)free_memory, (unsigned int)total_memory);

    /**
     *  Assign the CPU arrays:
     */
    for (int i = 0; i < array_length; i++) {
        cpu__v1[i] = i;
        cpu__v2[i] = 2 * i;
    }

    print_array("v1", cpu__v1, array_length);
    print_array("v2", cpu__v2, array_length);

    /**
     *  Copy the arrays from the CPU to the GPU (the gpu array goes first)
     */
    hipMemcpy(gpu__v1, cpu__v1, array_length * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(gpu__v2, cpu__v2, array_length * sizeof(float), hipMemcpyHostToDevice);

    /**
     *  Run the GPU kernel
     */
    gpu__vector_add<<<1, blockDimensions>>>(gpu__v1, gpu__v2, gpu__sum);

    /**
     *  Move the sum from the array on the GPU to the array on the CPU.
     */
    hipMemcpy(cpu__sum, gpu__sum, array_length * sizeof(float), hipMemcpyDeviceToHost);

    /**
     *  Print out the sum.
     */
    print_array("sum", cpu__sum, array_length);

    hipFree(gpu__v1);
    hipFree(gpu__v2);
    hipFree(gpu__sum);

    hipMemGetInfo(&free_memory, &total_memory);
    printf("free memory: %u, total memory: %u (after free)\n", (unsigned int)free_memory, (unsigned int)total_memory);

    free(cpu__v1);
    free(cpu__v2);
    free(cpu__sum);
}
